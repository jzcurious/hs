#include "hip/hip_runtime.h"
#include <torch/extension.h>


template <typename scalar_t>
using accessor_1d = torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits>;

template <typename scalar_t>
using accessor_2d = torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>;


template <typename scalar_t>
__forceinline__ __device__
void gpuAtomicAdd(scalar_t *acc_ptr, scalar_t part_val) {
    #if __CUDA_ARCH__ >= 700
        if constexpr (std::is_same_v<scalar_t, c10::Half>) {
            atomicAdd(
                reinterpret_cast<half*>(acc_ptr), 
                static_cast<half>(part_val)
            );
        }
        else {
            atomicAdd(acc_ptr, part_val);
        }
    #else
        if constexpr (std::is_same_v<scalar_t, float>) {
            atomicAdd(acc_ptr, part_val);
        }
        else {
            assert(false && "Not supported CUDA device.");
        }
    #endif
}


template <int batch_frag, int weight_cols_frag, int weight_rows_frag, typename scalar_t>
__global__ void linear_fwd_kern_smem(
    const accessor_2d<scalar_t> input,
    const accessor_2d<scalar_t> weight,
    const accessor_1d<scalar_t> bias,
    accessor_2d<scalar_t> output) {

    __shared__ scalar_t local_input[batch_frag][weight_cols_frag];          // 32: 128b .. 256b
    __shared__ scalar_t local_weight_t[weight_cols_frag][weight_rows_frag]; // 32: 128b .. 256b
    __shared__ scalar_t local_bias[weight_cols_frag];                       //  4: 16b  ..  32b
    __shared__ scalar_t local_output[batch_frag][weight_rows_frag];         // 16: 64b  .. 128b

    auto k = blockIdx.x * blockDim.x + threadIdx.x;
    auto i = blockIdx.y * blockDim.y + threadIdx.y;
    auto j = blockIdx.z * blockDim.z + threadIdx.z;

    auto l_k = threadIdx.x;
    auto l_i = threadIdx.y;
    auto l_j = threadIdx.z;

    auto batch_size = input.size(0);
    auto weight_rows = weight.size(0);
    auto weight_cols = weight.size(1);

    bool guard = k < batch_size and i < weight_cols and j < weight_rows;

    if (guard) {
        if (l_j == 0) {
            local_input[l_k][l_i] = input[k][i];
        }

        if (l_k == 0) {
            local_weight_t[l_i][l_j] = weight[j][i];
        }

        if (l_k == 0 and l_i == 0) {
            local_bias[l_j] = bias[j];
        }

        if (l_i == 0) {
            local_output[l_k][l_j] = 0;
        }
    }

    __syncthreads();

    if (guard) {
        auto part = local_input[l_k][l_i] * local_weight_t[l_i][l_j];

        if (i == 0) {
           part += local_bias[l_j];
        }
        
        gpuAtomicAdd(&local_output[l_k][l_j], part);
    }

    __syncthreads();

    if (guard and l_i == 0) {
        gpuAtomicAdd(&output[k][j], local_output[l_k][l_j]);
    }
}


template <int batch_frag, int weight_cols_frag, int weight_rows_frag, typename scalar_t>
__global__ void linear_bwd_kern_smem(
    const accessor_2d<scalar_t> input,
    const accessor_2d<scalar_t> weight,
    const accessor_2d<scalar_t> d_output,
    accessor_2d<scalar_t> d_input,
    accessor_2d<scalar_t> d_weight,
    accessor_1d<scalar_t> d_bias) {

    __shared__ scalar_t local_input[batch_frag][weight_cols_frag];            // 32: 128b .. 256b
    __shared__ scalar_t local_weight_t[weight_cols_frag][weight_rows_frag];   // 32: 128b .. 256b
    __shared__ scalar_t local_d_output[batch_frag][weight_rows_frag];         // 16:  64b .. 128b
    __shared__ scalar_t local_d_input[batch_frag][weight_cols_frag];          // 32: 128b .. 256b
    __shared__ scalar_t local_d_weight_t[weight_cols_frag][weight_rows_frag]; // 32: 128b .. 256b
    __shared__ scalar_t local_d_bias[weight_rows_frag];                       //  4:  16b ..  32b

    auto k = blockIdx.x * blockDim.x + threadIdx.x;
    auto i = blockIdx.y * blockDim.y + threadIdx.y;
    auto j = blockIdx.z * blockDim.z + threadIdx.z;

    auto l_k = threadIdx.x;
    auto l_i = threadIdx.y;
    auto l_j = threadIdx.z;

    auto batch_size = input.size(0);
    auto weight_rows = weight.size(0);
    auto weight_cols = weight.size(1);

    bool guard = k < batch_size and i < weight_cols and j < weight_rows;

    if (guard) {
        if (l_j == 0) {
            local_input[l_k][l_i] = input[k][i];
            local_d_input[l_k][l_i] = 0;
        }

        if (l_k == 0) {
            local_weight_t[l_i][l_j] = weight[j][i];
            local_d_weight_t[l_i][l_j] = 0;
        }
        
        if (l_i == 0) {
            local_d_output[l_k][l_j] = d_output[k][j];
        }

        if (l_k == 0 and l_i == 0) {
            local_d_bias[l_j] = 0;           
        }
    }

    __syncthreads();

    if (guard) {
        gpuAtomicAdd(&local_d_input[l_k][l_i],
            local_d_output[l_k][l_j] * local_weight_t[l_i][l_j]);
        
        gpuAtomicAdd(&local_d_weight_t[l_i][l_j],
            local_d_output[l_k][l_j] * local_input[l_k][l_i]);

        if (i == 0) {
            gpuAtomicAdd(&local_d_bias[l_j], local_d_output[l_k][l_j]);
        }
    }

    __syncthreads();

    if (guard) {
        if (l_j == 0) {
            gpuAtomicAdd(&d_input[k][i], local_d_input[l_k][l_i]);
        }

        if (l_k == 0) {
            gpuAtomicAdd(&d_weight[j][i], local_d_weight_t[l_i][l_j]);
        }

        if (l_k == 0 and l_i == 0) {
            gpuAtomicAdd(&d_bias[j], local_d_bias[l_j]);
        }
    }
}


#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_ARG(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define CHECK_COMPATIBILITY(x, y, d1, d2) \
    TORCH_CHECK_LINALG(x.size(d1) == y.size(d2), \
    #x " must be the same size by dim(" #d1 ") as " #y " by dim(" #d2 ")")


__forceinline__ unsigned int div_and_ceil(float x, float y) {
    return ceil(x / y);
}


torch::Tensor linear_forward(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor bias) {

    CHECK_ARG(input);
    CHECK_ARG(weight);
    CHECK_ARG(bias);

    CHECK_COMPATIBILITY(input, weight, 1, 1);
    CHECK_COMPATIBILITY(bias, weight, 0, 0);

    auto batch_size = input.size(0);
    auto weight_rows = weight.size(0);
    auto weight_cols = weight.size(1);

    auto output = torch::zeros(
        {batch_size, weight_rows},
        input.options()
    );

    constexpr dim3 block_dim = {4, 8, 4};

    const dim3 grid_dim = {
        div_and_ceil(batch_size, block_dim.x),
        div_and_ceil(weight_cols, block_dim.y),
        div_and_ceil(weight_rows, block_dim.z)
    };

    AT_DISPATCH_FLOATING_TYPES(
        input.scalar_type(),
        "linear_forward",
        ([&] {
            linear_fwd_kern_smem<block_dim.x, block_dim.y, block_dim.z><<<grid_dim, block_dim>>>(
                input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                weight.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                bias.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>()
            );
        })
    );

    return output;
}


std::vector<torch::Tensor> linear_backward(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor bias,
    torch::Tensor d_output) {

    CHECK_ARG(input);
    CHECK_ARG(weight);
    CHECK_ARG(bias);
    CHECK_ARG(d_output);

    CHECK_COMPATIBILITY(input, weight, 1, 1);
    CHECK_COMPATIBILITY(bias, weight, 0, 0);
    CHECK_COMPATIBILITY(d_output, weight, 1, 0);
    CHECK_COMPATIBILITY(d_output, bias, 1, 0);

    auto batch_size = input.size(0);
    auto weight_rows = weight.size(0);
    auto weight_cols = weight.size(1);

    auto d_input = torch::zeros_like(input);
    auto d_weight = torch::zeros_like(weight);
    auto d_bias = torch::zeros_like(bias);

    constexpr dim3 block_dim = {4, 8, 4};

    const dim3 grid_dim = {
        div_and_ceil(batch_size, block_dim.x),
        div_and_ceil(weight_cols, block_dim.y),
        div_and_ceil(weight_rows, block_dim.z)
    };

    AT_DISPATCH_FLOATING_TYPES(
        input.scalar_type(),
        "linear_backward",
        ([&] {
            linear_bwd_kern_smem<block_dim.x, block_dim.y, block_dim.z><<<grid_dim, block_dim>>>(
                input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                weight.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                d_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                d_input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                d_weight.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                d_bias.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>()
            );
        })
    );

    return {d_input, d_weight, d_bias};
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("linear_forward", &linear_forward, "Custom linear layer (forward)");
    m.def("linear_backward", &linear_backward, "Custom linear layer (backward)");
}