#include "hip/hip_runtime.h"
#include <torch/extension.h>


template <typename scalar_t>
__forceinline__ __device__
void gpuAtomicAdd(scalar_t *acc_ptr, scalar_t part_val) {
    #if __CUDA_ARCH__ >= 700
        if constexpr (std::is_same_v<scalar_t, c10::Half>) {
            atomicAdd(
                reinterpret_cast<half*>(acc_ptr), 
                static_cast<half>(part_val)
            );
        }
        else {
            atomicAdd(acc_ptr, part_val);
        }
    #else
        if constexpr (std::is_same_v<scalar_t, float>) {
            atomicAdd(acc_ptr, part_val);
        }
        else {
            assert(false && "Not supported CUDA device.");
        }
    #endif
}


template <typename scalar_t>
__global__ void linear_forward_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> input,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> weights,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> bias,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> output) {

    auto k = blockIdx.x * blockDim.x + threadIdx.x;
    auto i = blockIdx.y * blockDim.y + threadIdx.y;
    auto j = blockIdx.z * blockDim.z + threadIdx.z;

    bool guard = i < weights.size(0) and j < weights.size(1) and k < input.size(0);

    if (guard) {
        auto part = input[k][i] * weights[i][j];

        if (i == 0) {
           part += bias[j];
        }
        
        gpuAtomicAdd(&output[k][j], part);
    }
}


template <typename scalar_t>
__global__ void linear_backward_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> input,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> weights,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> bias,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_output,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_input,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_weights,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> d_bias) {

    auto k = blockIdx.x * blockDim.x + threadIdx.x;
    auto i = blockIdx.y * blockDim.y + threadIdx.y;
    auto j = blockIdx.z * blockDim.z + threadIdx.z;

    bool guard = i < weights.size(0) and j < weights.size(1) and k < input.size(0);

    if (guard) {
        gpuAtomicAdd(&d_input[k][i], d_output[k][j] * weights[i][j]);
        gpuAtomicAdd(&d_weights[i][j], d_output[k][j] * input[k][i]);

        if (i == 0) {
            gpuAtomicAdd(&d_bias[j], d_output[k][j]);
        }
    }
}


#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_ARG(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define CHECK_COMPATIBILITY(x, y, d1, d2) \
    TORCH_CHECK(x.size(d1) == y.size(d2), \
    #x " must be the same size by dim(" #d1 ") as " #y " by dim(" #d2 ")")


__forceinline__ unsigned int div_and_ceil(float x, float y) {
    return ceil(x / y);
}


__forceinline__ std::tuple<dim3, dim3> configure_grid(
    unsigned int nx, unsigned int ny, unsigned int nz) {

    const dim3 block_size = {4, 8, 4};

    const dim3 grid_size = {
        div_and_ceil(nx, block_size.x),
        div_and_ceil(ny, block_size.y),
        div_and_ceil(nz, block_size.z)
    };

    return {grid_size, block_size}; 
}


torch::Tensor linear_forward(
    torch::Tensor input,
    torch::Tensor weights,
    torch::Tensor bias) {

    CHECK_ARG(input);
    CHECK_ARG(weights);
    CHECK_ARG(bias);

    CHECK_COMPATIBILITY(input, weights, 1, 0);
    CHECK_COMPATIBILITY(bias, weights, 0, 1);

    auto output = torch::zeros({input.size(0), weights.size(1)}, input.options());

    dim3 grid_size, block_size;
    std::tie(grid_size, block_size) = configure_grid(
        input.size(0), input.size(1), weights.size(1));

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        input.type(),
        "linear_forward",
        ([&] {
            linear_forward_kernel<<<grid_size, block_size>>>(
                input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                weights.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                bias.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>()
            );
        })
    );

    return output;
}


std::vector<torch::Tensor> linear_backward(
    torch::Tensor input,
    torch::Tensor weights,
    torch::Tensor bias,
    torch::Tensor d_output) {

    CHECK_ARG(input);
    CHECK_ARG(weights);
    CHECK_ARG(bias);
    CHECK_ARG(d_output);

    CHECK_COMPATIBILITY(input, weights, 1, 0);
    CHECK_COMPATIBILITY(bias, weights, 0, 1);
    CHECK_COMPATIBILITY(d_output, weights, 1, 1);
    CHECK_COMPATIBILITY(d_output, bias, 1, 0);

    auto d_input = torch::zeros_like(input);
    auto d_weights = torch::zeros_like(weights);
    auto d_bias = torch::zeros_like(bias);

    dim3 grid_size, block_size;
    std::tie(grid_size, block_size) = configure_grid(
        input.size(0), input.size(1), weights.size(1));
    
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        input.type(),
        "linear_backward",
        ([&] {
            linear_backward_kernel<<<grid_size, block_size>>>(
                input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                weights.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                bias.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                d_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                d_input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                d_weights.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                d_bias.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>()
            );
        })
    );

    return {d_input, d_weights, d_bias};
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("linear_forward", &linear_forward, "Custom linear layer (forward)");
    m.def("linear_backward", &linear_backward, "Custom linear layer (backward)");
}