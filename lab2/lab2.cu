#include "hip/hip_runtime.h"
#include <torch/extension.h>


__global__ void d_add(float *a, float *b, float *c, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {
        c[i] = a[i] + b[i];
    }
}


#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define CHECK_SIZE(x, y) TORCH_CHECK(x.is_same_size(y), #y " must be the same size as " #x)

const int block_size = 128;


__forceinline__ int calc_grid_size(int m) {
    return (m + block_size - 1) / block_size;
}


torch::Tensor add(torch::Tensor a, torch::Tensor b) {
    CHECK_INPUT(a);
    CHECK_INPUT(b);
    CHECK_SIZE(a, b);

    auto c = torch::empty_like(a);
    int n = a.numel();

    d_add<<<calc_grid_size(n), block_size>>>(
        a.data_ptr<float>(),
        b.data_ptr<float>(),
        c.data_ptr<float>(),
        n
    );

    return c;
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("my_add", &add, "Custom vector addition");
}